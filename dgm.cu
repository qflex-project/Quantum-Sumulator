#include "hip/hip_runtime.h"
#include <iostream>
#include "dgm.h"
#include <omp.h>
#include <unistd.h>
#include <cstdio>
#include <iterator>
#include <bitset>
#include <random>

void Tokenize(const string& str, vector<string>& tokens, const string& delimiters = ",")
{
	// Skip delimiters at beginning.
	string::size_type lastPos = str.find_first_not_of(delimiters, 0);
	// Find first "non-delimiter".
	string::size_type pos = str.find_first_of(delimiters, lastPos);

	while (string::npos != pos || string::npos != lastPos)
	{
		// Found a token, add it to the vector.
		tokens.push_back(str.substr(lastPos, pos - lastPos));
		// Skip delimiters.  Note the "not_of"
		lastPos = str.find_first_not_of(delimiters, pos);
		// Find next "non-delimiter"
		pos = str.find_first_of(delimiters, lastPos);
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////

std::complex <float>* GenericExecute(std::complex <float> *state, string function, int qubits, int type, int threads, int factor = 0){
	DGM dgm;
	dgm.exec_type = type;
	dgm.n_threads = threads;
	dgm.qubits = qubits;
	dgm.factor = factor;

	dgm.setMemory(state);

	dgm.executeFunction(function);

	state = dgm.state;

	dgm.state = NULL;

	return state;
}

std::complex <float>* GenericExecute(std::complex <float> *state, vector<string> function, int qubits, int type, int threads, int factor = 0){
	DGM dgm;
	dgm.exec_type = type;
	dgm.n_threads = threads;
	dgm.qubits = qubits;
	dgm.factor = factor;
	dgm.setMemory(state);

	dgm.executeFunction(function);

	dgm.state = NULL;

	return state;
}

///////////////////////////////////////////////////////////////////////////////////////////////

DGM::DGM(){
	MAX_QB = QB_LIMIT;
	MAX_PT = PT_TAM;

	pts = NULL;
	state = NULL;
	en_print = false;
	exec_type = t_CPU;
	factor = 1;
	multi_gpu = 1;
}

DGM::~DGM(){erase();}

void DGM::setExecType(int type){
	exec_type = type;
}

void DGM::printPTs(){
	for (int i = 0; i < vec_pts.size() -1; i++){
		vec_pts[i]->print();
	}
}

void DGM::erase(){
	if (!pts) return;

	long i = 0;
	while (pts[i] != NULL){
		pts[i]->destructor();
		free(pts[i]);
		i++;
	}

	vec_pts.clear();
	pts = NULL;
}

void DGM::allocateMemory(){
	state = (std::complex <float>*) calloc(pow(2, qubits), sizeof(std::complex <float>));
}

void DGM::setMemory(std::complex <float>* mem){
	freeMemory();
	state = mem;
}

void DGM::freeMemory(){
	if (state) free(state);
	state = NULL;
}

void DGM::setMemoryValue(int pos){
	state[pos] = 1;
}

void DGM::setSuperposition(){
	auto value = sqrt(1.0/(pow(2, qubits)));
	for (long i = 0; i < pow(2, qubits); i++) state[i] = value;
}

int DGM::measure(int q_pos){
	long size = pow(2.0, qubits);

	long shift = (qubits - 1 - q_pos);
	long mask = 1 << shift;

	int count_one, count_zero, num_pb;
	float zero, one, norm_factor, r;
	one = zero = 0;

	//#pragma omp for;
	for (long i = 0; i < size; i++){
		if (i & mask)
			one += norm(state[i]);
		else
			zero += norm(state[i]);
	}

	long m;

	random_device rd;
	mt19937 gen(rd());
	uniform_real_distribution<> dis(0.05, 0.95);

	r = dis(gen);

	if (r > zero){
		norm_factor = sqrt(one);
		m = 1;
	}
	else{
		norm_factor = sqrt(zero);
		m = 0;
	}

	//#pragma omp for
	for (long i = 0; i < size/2; i++){
		long pos0 = (i * 2) - (i & (mask-1));
		long pos1 = pos0 | mask;

		//std::cout << "Pos0 " << pos0 << ": " << real(state[pos0]) << " + " << imag(state[pos0]) << "i" << endl;
		//std::cout << "Pos1 " << pos1 << ": " << real(state[pos1]) << " + " << imag(state[pos1]) << "i" << endl;

		if (m) {
			state[pos0]	= state[pos1]/norm_factor;
		}
		else {
			state[pos0]	= state[pos0]/norm_factor;
		}

		state[pos1] = 0.0;

		//std::cout << "Pos0 " << pos0 << ": " << real(state[pos0]) << " + " << imag(state[pos0]) << "i" << endl;
		//std::cout << "Pos1 " << pos1 << ": " << real(state[pos1]) << " + " << imag(state[pos1]) << "i" << endl;

		//cout << "###" << endl;
	}

	std::cout << q_pos << ": " << m << " (" << zero << " , " << one << ") " << r << " " << mask << " " << shift << " (" << sqrt(zero) << " , " << sqrt(one) << ") " << std::endl;

	return m;
}

void DGM::colapse(int q_pos, int value){
	long size = pow(2.0, qubits);

	long shift = (qubits - 1 - q_pos);
	long mask = 1 << shift;

	float norm_factor = 0;
	for (long i = 0; i < size; i++){
		if (((i >> shift) & 1) == value) {
			norm_factor += norm(state[i]);
		}
	}

	norm_factor = sqrt(norm_factor);
	int m = value;

	for (long i = 0; i < size/2; i++){
		long pos0 = (i * 2) - (i & (mask-1));
		long pos1 = pos0 | mask;

		if (m) {
			state[pos0]	= state[pos1]/norm_factor;
		}
		else {
			state[pos0]	= state[pos0]/norm_factor;
		}

		state[pos1] = 0.0;
	}

	std::cout << "colapse " << q_pos << " - " << value << " - " << norm_factor << std::endl;
}

void DGM::printProbability(int q_pos){
	long size = pow(2.0, qubits);

	long shift = (qubits - 1 - q_pos);
	long mask = 1 << shift;

	int count_one, count_zero, num_pb;
	float zero, one, norm_factor, r;
	one = zero = 0;

	//#pragma omp for;
	for (long i = 0; i < size; i++){
		if (i & mask)
			one += norm(state[i]);
		else
			zero += norm(state[i]);
	}

	if (std::abs(zero) < 1e-5) zero = 0;
	if (std::abs(one) < 1e-5) one = 0;

	std::cout << q_pos << " -- (" << zero << " , " << one << ")" << std::endl;
}


map <long, float> DGM::measure(vector<int> q_pos){
	long mask = 0;

	for (int i =0; i < q_pos.size(); i++) mask = mask | (1<<(qubits - 1 - q_pos[i]));

	map <long, float> m;

	long size = pow(2.0, qubits);

	for (long i =0; i < size; i++) m[i&mask] += pow(state[i].real(), 2.0) + pow(state[i].imag(), 2.0);

	return m;
}

void DGM::setFunction(string function, int it, bool er){
	vector <string> steps;

	Tokenize(function, steps, ";");

	setFunction(steps, it, er);
}

void DGM::setFunction(vector <string> steps, int it, bool er){
	if (er) erase();
	else vec_pts.pop_back();


	vector <PT*> step_pts, vec_tmp;
	map<long, Group> gps;

	for (long j = 0; j< it; j++)
	for (long i = 0; i < steps.size(); i++){
		gps = genGroups(steps[i]);
		genPTs(gps, step_pts);

		if (i%2)
			sort(step_pts.begin(), step_pts.end(), increasing);
		else
			sort(step_pts.begin(), step_pts.end(), decreasing);

		vec_pts.insert(vec_pts.end(), step_pts.begin(), step_pts.end());
	}

	vec_pts.push_back(NULL);

	pts = &vec_pts[0];
}

map <long, Group> DGM::genGroups(string step){
	vector <string> ops;
	Tokenize(step, ops); //separa os operadores usando "," como delimitador
	qubits = ops.size();

	size_t found_c, found_t, p;
	string str;
	long pos, ctrl_value, ctrl_num;
	
	map<long, Group> gps;

	char * pEnd;
	pos = 0;
	vector<string>::iterator it;
	for (it = ops.begin() ; it != ops.end(); ++it){ //percorre os operadores
		str = *it;
		//cout << str << endl;
		found_c = str.find("Control"); //tamanho 7
		found_t = str.find("Target");  //tamanho 6
		p = str.find("(") + 1;

		if (found_c != string::npos){ //Controle
			ctrl_num = strtol(str.c_str()+7, &pEnd, 10);
			ctrl_value = strtol(str.c_str()+p, &pEnd, 10);

			gps[ctrl_num].ctrl.push_back(ctrl_value); //adicona o valor do controle
			gps[ctrl_num].pos_ctrl.push_back(pos);  //e a sua posição ao map relacionado ao controle
		}
		else if(found_t != string::npos){ //Target
			ctrl_num = strtol(str.c_str()+6, &pEnd, 10);
			str = str.substr(p, str.size()-p-1);

			gps[ctrl_num].ops.push_back(str);     //adicona o operador
			gps[ctrl_num].pos_ops.push_back(pos); //e a sua posição ao map relacionado ao target
		}
		else{ //operador normal
			if (str != "ID"){ //se for ID ignora
				gps[0].ops.push_back(str);     //adiciona o operador
				gps[0].pos_ops.push_back(pos); //e a sua posição ao map '0'
			}
		}
		pos++;
	}
	
	return gps;
}

void DGM::genPTs(map<long, Group> &gps, vector <PT*> &step_pts){
	step_pts.clear();
	Gates gates;

	map<long,Group>::iterator it;	
	Group gp;
	PT* pt;
	long ctrl_mask, ctrl_value, ctrl_count;
	long size;
	
	for (it = gps.begin(); it != gps.end(); ++it){ //percorre os grupos
		gp = it->second;
		size = gp.ops.size();
		
		ctrl_count = gp.ctrl.size();
		ctrl_value = ctrl_mask = 0;

		for (long i = 0; i < ctrl_count; i++){ //gera a mascara e o valor do controle (em binario)
			gp.pos_ctrl[i] =  qubits - gp.pos_ctrl[i] - 1;
			ctrl_mask += (1 << gp.pos_ctrl[i]);
			if (gp.ctrl[i]) ctrl_value += (1 << gp.pos_ctrl[i]);
		}

		for (int p = 0; p < size; p++){
			
			pt = (PT*) malloc(sizeof(PT));
			pt->affected = false;

			pt->qubits = 1;
			pt->start = qubits - gp.pos_ops[p];
			pt->end = pt->start - 1;
			pt->mat_size = 2;
			
			pt->matrix = gates.getMatrix(gp.ops[p]);

			pt->ctrl_value = ctrl_value;
			pt->ctrl_mask = ctrl_mask;
			pt->ctrl_count = ctrl_count;

			if (ctrl_count){
				pt->ctrl_pos = (long*)malloc(sizeof(long) * ctrl_count);
				copy(gp.pos_ctrl.begin(), gp.pos_ctrl.end(), pt->ctrl_pos);
			}

			step_pts.push_back(pt);
		}
	}
}

void DGM::genMatrix(std::complex <float>* matrix, vector<std::complex <float>*> &matrices, long tam, long current, long line, long column, std::complex <float> cmplx){
	if (cmplx == COMPLEX_ZERO) return;

	if (current == tam){ //percorreu até a ultima matriz
		matrix[line*(1<<tam) + column] = cmplx;
		return;
	}

	for (long l = 0; l < 2; l++)
		for (long c = 0; c < 2; c++)
			genMatrix(matrix, matrices, tam, current+1, (line<<1)|l, (column<<1)|c, cmplx * matrices[current][l*2+c]);
}


void DGM::executeFunction(vector <string> function, int it){
	setFunction(function);
	execute(it);
}

void DGM::executeFunction(string function, int it){
	if (function == "") return;

	setFunction(function);
	execute(it);
}


std::complex <float>* DGM::execute(int it){
	std::complex <float>* result = state;

	switch (exec_type){
		case t_CPU:
			CpuExecution1(it);
			break;
		case t_PAR_CPU:
			PCpuExecution1(state, pts, qubits, n_threads, cpu_coales, cpu_region, it);
			break;
		#ifndef ONLY_CPU
		case t_GPU:
			result = GpuExecutionWrapper(state, pts, qubits, gpu_coales, gpu_region, multi_gpu, tam_block, rept, it);
			break;
		case t_HYBRID:
			HybridExecution2(pts);
			break;
		#endif
		default:
			cout << "Erro exec type" << endl;
			exit(1);
	}

	return result;
}


void DGM::CountOps(int it){
	dense = main_diag = sec_diag = c_dense = c_main_diag = c_sec_diag = 0;

	for (int i =0; pts[i]!=NULL; i++){
		long mt = pts[i]->matrixType();
		switch (mt){
			case DENSE:
				(pts[i]->ctrl_mask) ? c_dense++ : dense++;
				break;
			case DIAG_PRI:
				(pts[i]->ctrl_mask) ? c_main_diag++ : main_diag++;
				break;
			case DIAG_SEC:
				(pts[i]->ctrl_mask) ? c_sec_diag++ : sec_diag++;
				break;
			default:
				cout << "Error on operator type" << endl;
				exit(1);
		}
	}

	dense *= it;
	c_dense *= it;
	main_diag *= it;
	c_main_diag *= it;
	sec_diag *= it;
	c_sec_diag *= it;

	total_op = dense + c_dense + main_diag + c_main_diag + sec_diag + c_sec_diag;
}

void DGM::CpuExecution1(int it){
	long mem_size = pow(2.0, qubits);

	for (int x = 0; x < it; x++){
		long i = 0;
		while (pts[i] != NULL){
			long mt = pts[i]->matrixType();

			switch (mt){
				case DENSE:
					CpuExecution1_1(pts[i], mem_size);
					break;
				case DIAG_PRI:
					CpuExecution1_2(pts[i], mem_size);
					break;
				case DIAG_SEC:
					CpuExecution1_3(pts[i], mem_size);
					break;
				default:
					exit(1);
			}
			i++;
		}
	}
}

void DGM::CpuExecution1_1(PT *pt, long mem_size){ //Denso
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;
	
	std::complex <float> tmp;
		
	if (!pt->ctrl_count){ 			//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		long mask = ~(pt->ctrl_mask | shift);
		long inc = (~mask) + 1;

		for (long pos = 0; pos < mem_size; pos = (pos+inc) & mask){
			pos0 = pos | pt->ctrl_value;
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];			
			state[pos0] = tmp;
		}
	}
}

void DGM::CpuExecution1_2(PT *pt, long mem_size){ //Diagonal Principal
	long pos0, shift = pt->end;
		
	if (!pt->ctrl_count)	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++)
			state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
	else{					//operador controlado
		long mask = ~(pt->ctrl_mask);
		long inc = (~mask) + 1;

		for (long pos = 0; pos < mem_size; pos = (pos+inc) & mask){
			pos0 = pos | pt->ctrl_value;

			state[pos0] = pt->matrix[((pos0 >> shift) & 1) * 3] * state[pos0];
		}
	}
}

void DGM::CpuExecution1_3(PT *pt, long mem_size){ //Diagonal Secundária
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;

	std::complex <float> tmp;
		
	if (!pt->ctrl_count){ 	//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		long mask = ~(pt->ctrl_mask | shift);
		long inc = (~mask) + 1;
		
		for (long pos = 0; pos < mem_size; pos = (pos+inc) & mask){
			pos0 = pos | pt->ctrl_value;
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void DGM::CpuExecution2_1(PT *pt, long mem_size){ //Denso
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;
	mem_size /= 2;

	std::complex <float> tmp;
		
	if (!pt->ctrl_count) 			//operador não controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
			state[pos0] = tmp;
		}
	else{					//operador controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;
			if ((pos0 & pt->ctrl_mask) == pt->ctrl_value){
				tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
				state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];			
				state[pos0] = tmp;
			}
		}
	}
}

void DGM::CpuExecution2_2(PT *pt, long mem_size){ //Diagonal Principal
	long shift = pt->end;
		
	if (!pt->ctrl_count)	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++)
			state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
	else					//operador controlado
		for (long pos = 0; pos < mem_size; pos++)
			if ((pos & pt->ctrl_mask) == pt->ctrl_value)
				state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];

}



void DGM::CpuExecution2_3(PT *pt, long mem_size){ //Diagonal Secundária
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;
	mem_size /= 2;

	std::complex <float> tmp;
		
	if (!pt->ctrl_count) 	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	else					//operador controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;
			if ((pos0 & pt->ctrl_mask) == pt->ctrl_value){
				tmp = pt->matrix[1] * state[pos1];
				state[pos1] = pt->matrix[2] * state[pos0];
				state[pos0] = tmp;
			}
		}
	
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void DGM::CpuExecution3_1(PT *pt, long mem_size){ //Denso
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;

	std::complex <float> tmp;
		
	if (!pt->ctrl_count){ 			//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		vector <long> gap, max;
		long i, c, mask;

		mask = pt->ctrl_mask | shift;

		c = 0;
		for (i = 0; i < qubits; i++){
			if (((mask >> i) & 1) == 0) c++;
			else if (c){
				gap.push_back(1<<(i-c));
				max.push_back(1<<i);
				c = 0;
			}
		}
		if (c){
			gap.push_back(1<<(i-c));
			max.push_back(1<<(qubits+1));
		}
		else{	
			gap.push_back(1<<(qubits+1));
			max.push_back(1<<(qubits+2));
		}

		long pos = 0;

		while (pos < mem_size){
				pos0 = pos | pt->ctrl_value;
				pos1 = pos0 | shift;

				//cout << pos0 <<  " " << pos1 << endl; 

				tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
				state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];			
				state[pos0] = tmp;

				pos += gap[0];
				i = 0;
				while (pos & max[i]){
					pos ^= max[i++];
					pos += gap[i];
				}

		}
		//cout << endl;
	}	
}

void DGM::CpuExecution3_2(PT *pt, long mem_size){ //Diagonal Principal
	long pos0, shift = pt->end;
		
	if (!pt->ctrl_count)	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++)
			state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
	else{					//operador controlado
		vector <long> gap, max;
		long i, c, mask;

		mask = pt->ctrl_mask;

		c = 0;
		for (i = 0; i < qubits; i++){
			if (((mask >> i) & 1) == 0) c++;
			else if (c){
				gap.push_back(1<<(i-c));
				max.push_back(1<<i);
				c = 0;
			}
		}
		if (c){
			gap.push_back(1<<(i-c));
			max.push_back(1<<(qubits+1));
		}
		else{	
			gap.push_back(1<<(qubits+1));
			max.push_back(1<<(qubits+2));
		}

		long pos = 0;

		while (pos < mem_size){
				pos0 = pos | pt->ctrl_value;

				//cout << pos0 << endl; 
				state[pos0] = pt->matrix[((pos0 >> shift) & 1) * 3] * state[pos0];

				pos += gap[0];
				i = 0;
				while (pos & max[i]){
					pos ^= max[i++];
					pos += gap[i];
				}

		}
	}
}

void DGM::CpuExecution3_3(PT *pt, long mem_size){ //Diagonal Secundária
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;


	std::complex <float> tmp;
		
	if (!pt->ctrl_count){ 	//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		vector <long> gap, max;
		long i, c, mask;

		mask = pt->ctrl_mask | shift;

		c = 0;
		for (i = 0; i < qubits; i++){
			if (((mask >> i) & 1) == 0) c++;
			else if (c){
				gap.push_back(1<<(i-c));
				max.push_back(1<<i);
				c = 0;
			}
		}
		if (c){
			gap.push_back(1<<(i-c));
			max.push_back(1<<(qubits+1));
		}
		else{	
			gap.push_back(1<<(qubits+1));
			max.push_back(1<<(qubits+2));
		}

		long pos = 0;

		while (pos < mem_size){
			pos0 = pos | pt->ctrl_value;
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;

			pos += gap[0];
			i = 0;
			while (pos & max[i]){
				pos ^= max[i++];
				pos += gap[i];
			}
		}
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void PCpuExecution1(std::complex <float> *state, PT **pts, int qubits, long n_threads, int coales, int region, int it){
	long i, start, end;
	i = start = 0;
	while (pts[i] != NULL){
		long count = coales;
		long reg_mask = (coales)? (1 << coales) - 1 : 0;

		//Pega os operadores que estão dentro da região coalescida (reg_mask inicial),
		//e acrescenta operadores em qubits fora dela até chegar ao limite da região (region definida)
		start = i;
		while (count < region && pts[i] != NULL){					//Repete enquanto o número de qubits da região não atingir o limite (region) e houver operadores
			if (//pts[i]->matrixType() != DIAG_PRI &&					//O qubit de operadores de diagonal principal não importa para região (sempre podem ser acrescentados)
				!((reg_mask >> pts[i]->end) & 1)){				//Se o qubit do operador estiver fora da região (reg_mask), incrementa o contador de qubits da região
				count++;
			}

			if (count <= region)// && pts[i]->matrixType() != DIAG_PRI)
				reg_mask = reg_mask | (1 << pts[i]->end);			//Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)
				
			i++;
		}
		//Segue acerscentado até encontrar um operador que não esteja dentro da região
		while (pts[i] != NULL){
			if (((reg_mask >> pts[i]->end) & 1))// || pts[i]->matrixType() == DIAG_PRI)
				i++;
			else
				break;
		}
		end = i;													//Executa até o operador na posiçao 'i' (exclusive) nesta iteração


		//Se o número de qubits na região (count) não tiver atingido o limite (region),
		//acrescenta os ultimos qubits (final da mascara) à região até completar
		//for (long a = 1<<(qubits-1); count < region; a = a >> 1){
		for (long a = 1; count < region; a = a << 1){
			if (a & ~reg_mask){
				reg_mask = reg_mask | a;
				count++;
			}
		}

		if (count < region)
			region = count;

		long reg_count = (1 << (qubits - region)) + 1; 				//Número de regiões 			-	 +1 para a condição de parada incluir todos
		long pos_count = 1 << (region - 1); 						//Número de posições na região 	-	 -1 porque são duas posições por iteração

		omp_set_num_threads(n_threads);

		long ext_reg_id = 0;	//contador 'global' do número de regiões já computadas

		#pragma omp parallel
		{

			long reg_id;		//indentificador local da região

			//Define a primeira região (reg_id) da thread
			#pragma omp critical (teste)
			{
				reg_id = ext_reg_id;
				ext_reg_id = (ext_reg_id + reg_mask + 1) & ~reg_mask;
				reg_count--;
				if (reg_count <= 0)
					reg_id = -1;
			}

			int print = (omp_get_thread_num()==0);
			
			
			while (reg_id != -1){		
				//Computa os operadores
				PCpuExecution1_0(state, pts, qubits, start, end, pos_count, reg_id, reg_mask);
		
				//Define a próxima região (reg_id) da thread
				#pragma omp critical (teste)
				{
					reg_id = ext_reg_id;
					ext_reg_id = (ext_reg_id + reg_mask + 1) & ~reg_mask;
					reg_count--;
					if (reg_count <= 0)
						reg_id = -1;
				}
			}
		}
	}
}

void PCpuExecution1_0(std::complex <float> *state, PT **pts, int qubits, int start, int end, int pos_count, int reg_id, int reg_mask){
	PT *QG;
	long pos0, pos1;
	std::complex <float> tmp;

	//cout << "\nExecution" << endl;
	//cout << "reg_id " << getBinaryString(reg_id, qubits, true) << endl;
	//cout << "reg_mask " << getBinaryString(reg_mask, qubits, true) << endl;
	//cout << "pos_count " << pos_count << endl;
	//cout << "start " << start << endl;
	//cout << "end " << end << endl;

	for (int op = start; op < end; op++){
		QG = pts[op];
		long shift = (1 << QG->end);						//mascara com a posição do qubit do operador
		long mt = QG->matrixType();
		//if (mt == DIAG_PRI) shift = coalesc;	//se for um operador de diagonal principal, a posição do qubit não é relevante
		long pos_mask = reg_mask & ~shift;			//mascara da posição --- retira o 'shift' da reg_mask, para o 'inc pular sobre ' esse bit também
		long inc = ~pos_mask + 1;						  	//usado para calcular a proxima posição de uma região
		long pos = 0;

		//cout << "OP " << op << endl;
		//cout << "pos_mask " << getBinaryString(pos_mask, qubits, true) << endl;
		//cout << "shift " << getBinaryString(shift, qubits, true) << endl;
		//cout << "inc " << getBinaryString(inc, qubits, true) << endl;
					
		if (!QG->ctrl_count){
			switch (mt){
				case DENSE:
					for (long p = 0; p < pos_count; p++){
						pos0 = pos | reg_id;
						pos1 = pos0 | shift;
						pos = (pos+inc) & pos_mask;

						//cout << getBinaryString(pos0, qubits, true) << " - " << getBinaryString(pos1, qubits, true) << endl;

						tmp 		= QG->matrix[2] * state[pos0] + QG->matrix[3] * state[pos1];
						state[pos0] = QG->matrix[0] * state[pos0] + QG->matrix[1] * state[pos1];
						state[pos1] = tmp;
					}
					break;
				case DIAG_PRI:
					for (long p = 0; p < pos_count; p++){
							pos0 = pos | reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp			= QG->matrix[3] * state[pos1];
							state[pos0] *= QG->matrix[0];// * state[pos0];
							state[pos1] = tmp;// * state[pos1];tmp;
					}
					break;
				
				case DIAG_SEC:
					for (long p = 0; p < pos_count; p++){
							pos0 = pos | reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp 		= QG->matrix[2] * state[pos0];
							state[pos0] = QG->matrix[1] * state[pos1];
							state[pos1] = tmp;
					}
					break;
				default:
					printf("Erro de Tipo\n");
			}
		}
		//Importante: reg_id é o identificador da região e corresponde ao valor dos qubits externos à região de operação (reg_mask)
		else {			
			if ((QG->ctrl_mask & reg_id & ~reg_mask) == (QG->ctrl_value & ~reg_mask)){		//Verifica se a parte 'global' do controle satisfaz a região (reg_id)

				// É preciso arrumar o reg_mask retirando os qubits de controle que estão dentro da região e arrumar o reg_id para incluir o valor dos controles
				long ctrl_reg_id = reg_id | QG->ctrl_value;				//Esta operação inclui o valor dos controles locais no reg_id (funciona pois os valores globais já deram match)
				long ctrl_reg_mask = reg_mask;							//Valor inicial da mascara da região com controle
				long ctrl_pos_count = pos_count;						//Número inicial de posições a serem calculadas

				for (int i = 0, m = 1; i < qubits; i++, m = m << 1){ 	//percorre os qubits
					if (m & reg_mask & QG->ctrl_mask){					//se o qubit pertencer a região e for um controle:
						ctrl_reg_mask ^= m;								//	remove ele da região(reg_mask) (para não iterar sobre ele)
						ctrl_pos_count /= 2;							//	diminui a quantidade de posições que é preciso calcular.
					}
				}

				pos_mask = ctrl_reg_mask & ~shift;						//mascara da posição --- retira o 'shift' da reg_mask, para o 'inc pular sobre' esse bit também
				inc = ~pos_mask + 1;

				switch (mt){
					case DENSE:
						for (long p = 0; p < ctrl_pos_count; p++){
							pos0 = pos | ctrl_reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp 		= QG->matrix[2] * state[pos0] + QG->matrix[3] * state[pos1];
							state[pos0] = QG->matrix[0] * state[pos0] + QG->matrix[1] * state[pos1];
							state[pos1] = tmp;
						}
						break;
					case DIAG_PRI:
						for (long p = 0; p < ctrl_pos_count; p++){
							pos0 = pos | ctrl_reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp			= QG->matrix[3] * state[pos1];
							state[pos0] *= QG->matrix[0];
							state[pos1] = tmp;
						}
						break;
					
					case DIAG_SEC:
						for (long p = 0; p < ctrl_pos_count; p++){
							pos0 = pos | ctrl_reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp 		= QG->matrix[2] * state[pos0];
							state[pos0] = QG->matrix[1] * state[pos1];
							state[pos1] = tmp;
						}
						break;

					default:
						printf("Erro de Tipo");
				}
			}
		}
	}
}

#ifndef ONLY_CPU

void DGM::HybridExecution(PT **pts){
	long mem_size = pow(2.0, qubits);
	long qubits_limit = 20;
	long global_coales = 15; //(cpu_coales > gpu_coales) ? cpu_coales : gpu_coales;

	long global_region = qubits_limit;
	long global_start, global_end;

	long global_count, global_reg_mask, global_reg_count, global_pos_count, ext_proj_id; 

	omp_set_num_threads(n_threads);

	int i = 0;
	while (pts[i] != NULL){
		global_count = global_coales;
		global_reg_mask = (global_coales)? (1 << global_coales) - 1 : 0;

		//Realiza a projeção dos operadores de acordo com o limite de qubits que podem ser executados
		global_start = i;
		while (global_count < global_region && pts[i] != NULL){			//Repete enquanto o número de qubits da região não atingir o limite (region) e houver operadores
			if (//pts[i]->matrixType() != DIAG_PRI &&					//O qubit de operadores de diagonal principal não importa para região (sempre podem ser acrescentados)
			!((global_reg_mask >> pts[i]->end) & 1)){				
				global_count++;
			}

			if (global_count <= global_region)// && pts[i]->matrixType() != DIAG_PRI)
				global_reg_mask = global_reg_mask | (1 << pts[i]->end);			//Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)	

			i++;
		}

		while (pts[i] != NULL){
			if (((global_reg_mask >> pts[i]->end) & 1))// || pts[i]->matrixType() == DIAG_PRI)
				i++;
			else
				break;
		}
		global_end = i;

		//Se o número de qubits na região (count) não tiver atingido o limite (region),
		//acrescenta os ultimos qubits (final da mascara) à região até completar
		//for (long a = 1<<(qubits-1); count < region; a = a >> 1){
		for (long a = 1; global_count < global_region; a = a << 1){
			if (a & ~global_reg_mask){
				global_reg_mask = global_reg_mask | a;
				global_count++;
			}
		}

		if (global_count < global_region)
			global_region = global_count;
	
		global_reg_count = (1 << (qubits - global_region)) + 1; 				//Número de regiões	- +1 para a condição de parada incluir todos
		global_pos_count = 1 << (global_region - 1);

		/////////////////////////////////////////////////////////////////////////////////////////////////////

		ext_proj_id = 0;	//contador 'global' do número de regiões já computadas

		//Define a primeira região (reg_id) da thread

		#pragma omp parallel num_threads(n_threads)
		{
			if (omp_get_thread_num() < n_threads){  //CPU EXECUTION
				long cpu_proj_id;
				
				#pragma omp critical (global_hybrid)
				{
					cpu_proj_id = ext_proj_id;
					ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
					global_reg_count--;
					if (global_reg_count <= 0)
						cpu_proj_id = -1;
				}
	
				while (cpu_proj_id != -1){
					long cpu_i, cpu_start, cpu_end;

					cpu_start = global_start;
			
					cpu_i = cpu_start;
			
					while (cpu_start < global_end){
						long cpu_count = cpu_coales;
						long cpu_reg_mask = (cpu_coales)? (1 << cpu_coales) - 1 : 0;
			
						while ((cpu_count < cpu_region) && (cpu_i < global_end)){	//Tem que pertencer a região 'global'
							if (!((cpu_reg_mask >> pts[cpu_i]->end) & 1)){			//Se o qubit do operador estiver fora da região (reg_mask), incrementa o contador de qubits da região
								cpu_count++;
							}
		
							if (cpu_count <= cpu_region)// && pts[i]->matrixType() != DIAG_PRI)
								cpu_reg_mask = cpu_reg_mask | (1 << pts[cpu_i]->end);	//Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)
						
							cpu_i++;
						}
			
						while (cpu_i < global_end){
							if (((cpu_reg_mask >> pts[cpu_i]->end) & 1))// || pts[i]->matrixType() == DIAG_PRI)
								cpu_i++;
							else
								break;
						}
						cpu_end = cpu_i;
			
						for (long a = 1; cpu_count < cpu_region; a = a << 1){
							if ((a & global_reg_mask) && (a & ~cpu_reg_mask)){ //tem que não estar na região da cpu e estar na global
								cpu_reg_mask = cpu_reg_mask | a;
								cpu_count++;
							}
						}
	
						long cpu_reg_count = (1 << (global_region - cpu_region)) + 1; 		//Número de regiões				-	+1 para a condição de parada incluir todos
						long cpu_pos_count = 1 << (cpu_region - 1); 						//Número de posições na região	-	-1 porque são duas posições por iteração

				
						long cpu_ext_proj_id = 0;
						long inc_ext_proj_id = ~(cpu_reg_mask ^ global_reg_mask) & ((1 << qubits) - 1);
			
						long proj_id;		//indentificador local da região
						proj_id = cpu_ext_proj_id | cpu_proj_id;
						cpu_ext_proj_id = (cpu_ext_proj_id + inc_ext_proj_id + 1) & ~inc_ext_proj_id;
						cpu_reg_count--;
						
						while (proj_id != -1){
							//Computa os operadores
							PCpuExecution1_0(state, pts, qubits, cpu_start, cpu_end, cpu_pos_count, proj_id, cpu_reg_mask);
				
							proj_id = cpu_ext_proj_id | cpu_proj_id;
							cpu_ext_proj_id = (cpu_ext_proj_id + inc_ext_proj_id + 1) & ~inc_ext_proj_id;
							cpu_reg_count--;
							if (cpu_reg_count <= 0)
								proj_id = -1;
						}
			
						cpu_start = cpu_end;
					}
		
					#pragma omp critical (global_hybrid)
					{
						cpu_proj_id = ext_proj_id;
						ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
						global_reg_count--;
							if (global_reg_count <= 0)
						cpu_proj_id = -1;
					}
				}
				
			}
			//#pragma omp section          //GPU EXECUTION
			else{
				#ifndef ONLY_CPU
					long gpu_proj_id;
					
					#pragma omp critical (global_hybrid)
					{
						gpu_proj_id = ext_proj_id;
						ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
						global_reg_count--;
						if (global_reg_count <= 0)
							gpu_proj_id = -1;
					}

					while (gpu_proj_id != -1){
						//Project Gates
						vector <PT*> gpu_pts;
						
						int gpu_i;

						int map_qb[qubits];
						memset(map_qb, -1, qubits * sizeof(int));
			
						int m = 0;
						for (gpu_i = 0; gpu_i < qubits; gpu_i++){
							if ((1 << gpu_i) & global_reg_mask){
								map_qb[gpu_i] = m++;
							}
						}
						
						PT *aux;
						gpu_pts.clear();
						for (int gpu_i = global_start; gpu_i < global_end; gpu_i++){
							
							//verifica se o controle do operador satisfaz a parte global da região
							if ((pts[gpu_i]->ctrl_mask & gpu_proj_id & ~global_reg_mask) == (pts[gpu_i]->ctrl_value & ~global_reg_mask)){
								aux = new PT();

								aux->qubits = pts[gpu_i]->qubits;

								aux->matrix = pts[gpu_i]->matrix;
								aux->mat_size = pts[gpu_i]->mat_size;
								aux->ctrl_mask = pts[gpu_i]->ctrl_mask & global_reg_mask;
								aux->ctrl_value = pts[gpu_i]->ctrl_value & global_reg_mask;

								aux->end = map_qb[pts[gpu_i]->end];
								aux->start = aux->end - log2(aux->mat_size);

								aux->ctrl_count = 0;
								for (int c = global_coales; c < qubits; c++){
									if (aux->ctrl_mask & (1<<c)){
										aux->ctrl_count++;

										aux->ctrl_mask &= ~(1<<c);			//retira da mascara o controle do qubit atual (c)
										aux->ctrl_mask |= (1 << map_qb[c]);	//e coloca o qubit que ele mapeia (map_qb[c])

										if (aux->ctrl_value & (1<<c)){ 		//se o valor do controle for zero faz a mesma coisa para ctrl_value;
											aux->ctrl_mask &= ~(1<<c);
											aux->ctrl_mask |= (1 << map_qb[c]);
										}
									}
								}	

								gpu_pts.push_back(aux);
							}
						}
						gpu_pts.push_back(NULL);
						////////////////

						ProjectState(state, qubits, global_region, gpu_proj_id, global_reg_mask, multi_gpu);

						GpuExecutionWrapper(NULL, &gpu_pts[0], global_region, gpu_coales, gpu_region, multi_gpu, tam_block, rept, 1);
		
						GetState(state, qubits, global_region, gpu_proj_id, global_reg_mask, multi_gpu);

						for (int c = 0; c < gpu_pts.size() - 1; c++){
							delete gpu_pts[c];
						}
			
						#pragma omp critical (global_hybrid)
						{
							gpu_proj_id = ext_proj_id;
							ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
							global_reg_count--;
							if (global_reg_count <= 0)
								gpu_proj_id = -1;
						}
					}
				#endif
			}
		//}
		}
	}
}

void DGM::HybridExecution2(PT **pts){
	long mem_size = pow(2.0, qubits);
	long qubits_limit = 20;
	long global_coales = 11; //(cpu_coales > gpu_coales) ? cpu_coales : gpu_coales;

	long global_region = qubits_limit;
	long global_start, global_end;

	long global_count, global_reg_mask, global_reg_count, global_pos_count, ext_proj_id;

	cout << "HybridExecution2" << endl;
	cout << "n_threads: " << n_threads << endl;
	cout << "qubits: " << qubits << endl;
	cout << "global_region: " << global_region << endl;
	cout << "global_coales: " << global_coales << endl;
	cout << "cpu_coales: " << cpu_coales << endl;
	cout << "cpu_region: " << cpu_region << endl;

	omp_set_num_threads(n_threads);

	Barrier cpu_barrier(n_threads);

	while (pts[global_start] != NULL){		
		Projection global_proj;
		global_proj.setData(nullptr, pts, global_start, -1, qubits, global_coales, global_region);

		global_end = global_start + global_proj.operators_count;

		//cout << "\n#######\nGLOBAL PROJ " << global_start << " - " << global_end << endl;
		//global_proj.printInfo();
		//cout << "#######" << endl;

		long global_cpu_proj_id;

		Projection cpu_proj;
		long cpu_start, cpu_end, cpu_pos_count;

		#pragma omp parallel num_threads(n_threads)
		{
			long thread_id = omp_get_thread_num();
			//cout << string("ThreadID " + std::to_string(thread_id) + "\n");
			if (thread_id < n_threads){  //CPU EXECUTION		
				// get first global projection id	
				if (thread_id == 0){ // master thread for cpu
					global_cpu_proj_id = global_proj.getNextProjectionId();
					cpu_end = global_start; // set start operator

					//cout << string("\n#######\nFIRST GLOBAL CPU PROJ ID: " + getBinaryString(global_cpu_proj_id, qubits, true) + " - " + std::to_string(global_proj.count) + "\n");
				}

				// wait for all cpu threads
				cpu_barrier.arrive_and_wait(1);
	
				while (global_cpu_proj_id != -1){
					cpu_barrier.arrive_and_wait(2);
					// get first cpu sub projection from global projection

					while (cpu_end < global_end) {
						cpu_barrier.arrive_and_wait(3);
						if (thread_id == 0){ // master thread for cpu
							cpu_start = cpu_end;
							cpu_proj.setData(&global_proj, pts, cpu_start, global_end, qubits, cpu_coales, cpu_region);
							cpu_end = cpu_start + cpu_proj.operators_count;

							cpu_pos_count = 1 << (cpu_proj.region_size);

							//cout << string("\n#######\nSUB CPU PROJ " + std::to_string(cpu_start) + " - " + std::to_string(cpu_end) + "\n");
							//cpu_proj.printInfo();
						}
						cpu_barrier.arrive_and_wait(4);
						////////////////////////////////////////////////////
						long cpu_proj_id = cpu_proj.getNextProjectionId();
						
						while (cpu_proj_id != -1){
							//cout << string("ThreadId " + std::to_string(thread_id) + ": cpu_proj_id " + getBinaryString(cpu_proj_id, qubits, true) + " - " + std::to_string(cpu_proj.count) + "\n");
							//Computa os operadores
							PCpuExecution1_0(state, pts, qubits, cpu_start, cpu_end, cpu_pos_count / 2, cpu_proj_id, cpu_proj.region_mask);

							cpu_proj_id = cpu_proj.getNextProjectionId();
						}

						//cpu_barrier.arrive_and_wait();

						////////////////////////////////////////////////////
					}

					cpu_barrier.arrive_and_wait(5);
					// get next global projection id
					if (thread_id == 0){ // master thread for cpu
						global_cpu_proj_id = global_proj.getNextProjectionId();
						cpu_end = global_start; // reset start operator

						//cout << string("\n#######\nGLOBAL CPU PROJ ID " + getBinaryString(global_cpu_proj_id, qubits, true) + " - " + std::to_string(global_proj.count) + "\n");
					}
					cpu_barrier.arrive_and_wait(6);

				}
			}
			//#pragma omp section          //GPU EXECUTION
			else{
				cout << "\n#######\nGPU EXECUTION" << endl;
				long gpu_proj_id = global_proj.getNextProjectionId();

				while (gpu_proj_id != -1){
					//Project Gates
					vector <PT*> gpu_pts;
					
					int gpu_i;

					int map_qb[qubits];
					memset(map_qb, -1, qubits * sizeof(int));
		
					int m = 0;
					for (gpu_i = 0; gpu_i < qubits; gpu_i++){
						if ((1 << gpu_i) & global_reg_mask){
							map_qb[gpu_i] = m++;
						}
					}
					
					PT *aux;
					gpu_pts.clear();
					for (int gpu_i = global_start; gpu_i < global_end; gpu_i++){
						
						//verifica se o controle do operador satisfaz a parte global da região
						if ((pts[gpu_i]->ctrl_mask & gpu_proj_id & ~global_reg_mask) == (pts[gpu_i]->ctrl_value & ~global_reg_mask)){
							aux = new PT();

							aux->qubits = pts[gpu_i]->qubits;

							aux->matrix = pts[gpu_i]->matrix;
							aux->mat_size = pts[gpu_i]->mat_size;
							aux->ctrl_mask = pts[gpu_i]->ctrl_mask & global_reg_mask;
							aux->ctrl_value = pts[gpu_i]->ctrl_value & global_reg_mask;

							aux->end = map_qb[pts[gpu_i]->end];
							aux->start = aux->end - log2(aux->mat_size);

							aux->ctrl_count = 0;
							for (int c = global_coales; c < qubits; c++){
								if (aux->ctrl_mask & (1<<c)){
									aux->ctrl_count++;

									aux->ctrl_mask &= ~(1<<c);			//retira da mascara o controle do qubit atual (c)
									aux->ctrl_mask |= (1 << map_qb[c]);	//e coloca o qubit que ele mapeia (map_qb[c])

									if (aux->ctrl_value & (1<<c)){ 		//se o valor do controle for zero faz a mesma coisa para ctrl_value;
										aux->ctrl_mask &= ~(1<<c);
										aux->ctrl_mask |= (1 << map_qb[c]);
									}
								}
							}	

							gpu_pts.push_back(aux);
						}
					}
					gpu_pts.push_back(NULL);
					////////////////

					ProjectState(state, qubits, global_region, gpu_proj_id, global_reg_mask, multi_gpu);

					GpuExecutionWrapper(NULL, &gpu_pts[0], global_region, gpu_coales, gpu_region, multi_gpu, tam_block, rept, 1);
	
					GetState(state, qubits, global_region, gpu_proj_id, global_reg_mask, multi_gpu);

					for (int c = 0; c < gpu_pts.size() - 1; c++){
						delete gpu_pts[c];
					}
		
					gpu_proj_id = global_proj.getNextProjectionId();
				}
			}
		//}
		}

		global_start = global_end;
	}
}

#endif

void DGM::setCpuStructure(long cpu_region, long cpu_coales){
	this->cpu_region = cpu_region;
	this->cpu_coales = cpu_coales;
}

void DGM::setGpuStructure(long gpu_region, long gpu_coales, int rept){
	this->gpu_region = gpu_region;
	this->gpu_coales = gpu_coales;
	this->rept = rept;
	this->tam_block = 1 << gpu_region / 2 / rept;
}

void Projection::setData(Projection *parent_proj, PT **pts, long start, long end, long qubits, long coales, long region_size, bool include_main_diag) {
	region_mask = (coales > 0) ? (1 << coales) - 1 : 0; // mascara da região, inicializada com os quibts de coalescencia
	
	long count = coales; // conta o número de qubits na região

	//Realiza a projeção dos operadores de acordo com o limite de qubits que podem ser executados

	long i = start;
	while (count < region_size && (end < 0 || i < end) && pts[i] != NULL){	//Repete enquanto o número de qubits da região não atingir o limite (region) e houver operadores
		if (!(include_main_diag && pts[i]->matrixType() == DIAG_PRI)) { // verifica se pode incluir operadores de diagonal principal automaticamente, pois em alguns cenários eles não importam para a projeção e sempre podem ser acrescentados
			if (!((region_mask >> pts[i]->end) & 1)) { // se o qubit do operador não estiver sido acrescentado ainda, então incrementa o contador de qubits da região
				count++;

				if (count <= region_size) // se não tiver atingido o limite
					region_mask = region_mask | (1 << pts[i]->end);			// Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)
			}
		}

		i++;
	}

	// segue acrescentando operadores enquanto eles estiverem dentro da região
	while ((end < 0 || i < end) && pts[i] != NULL){
		if ((include_main_diag && pts[i]->matrixType() == DIAG_PRI) || ((region_mask >> pts[i]->end) & 1))
			i++;
		else
			break;
	}

	//Se o número de qubits na região (count) não tiver atingido o limite (region_size),
	//acrescenta os ultimos qubits (final da mascara) à região até completar
	//for (long a = 1<<(qubits-1); count < region; a = a >> 1){
	for (long b = 1; count < region_size; b = b << 1){
		if ((b & ~region_mask) && (!parent_proj || (b & parent_proj->region_mask))){
			region_mask = region_mask | b;
			count++;
		}
	}

	this->qubits = qubits;
	this->region_size = region_size;
	this->operators_count = i - start;
	this->count = 0;
	this->cur_id = 0;
	this->next_id = 0;

	if (parent_proj) {
		this->total = (1 << (parent_proj->region_size - region_size));

		this->parent_proj_id = parent_proj->cur_id;
		this->inc_mask = (region_mask | (~parent_proj->region_mask)) & ((1 << qubits) - 1);
	}
	else {
		this->total = (1 << (qubits - region_size));

		this->parent_proj_id = 0;
		this->inc_mask = region_mask;
	}
}

long Projection::getNextProjectionId() {
	std::lock_guard<std::mutex> lock(mutex_);

	count++;
	if (count > total)
		return -1;

	cur_id = next_id;
	next_id = ((cur_id + inc_mask + 1) & ~inc_mask); // & ((1 << qubits) - 1);

	return cur_id | parent_proj_id;
}

void Projection::printInfo() {
	cout << "Projection Info" << endl;
	cout << "region_size: " << region_size << endl;
	cout << "operators_count: " << operators_count << endl;
	cout << "count: " << count << endl;
	cout << "total: " << total << endl;
	cout << "cur_id: " << getBinaryString(cur_id, qubits, true) << endl;
	cout << "inc_mask: " << getBinaryString(inc_mask, qubits, true) << endl;
	cout << "region_mask: " << getBinaryString(region_mask, qubits, true) << endl;
	cout << "parent_proj_id: " << getBinaryString(parent_proj_id, qubits, true) << endl;
}

std::string getBinaryString(long num, int n, bool includeNum) {
    if (n < 1) {
        return "Invalid number of bits";
    }

    std::bitset<sizeof(long) * 8> bits(num); // Create a bitset with the integer value

    // Convert the bitset to a string
    std::string binaryString = bits.to_string();

    // Ensure the string has 'n' bits
    if (binaryString.length() < n) {
        // Pad the string with leading zeros
        binaryString = std::string(n - binaryString.length(), '0') + binaryString;
    } else if (binaryString.length() > n) {
        // Trim excess bits if the string has more than 'n' bits
        binaryString = binaryString.substr(binaryString.length() - n);
    }

	if (includeNum)
		binaryString = std::to_string(num) + " (" + binaryString + ")";

    return binaryString;
}

void report_num_threads(int level){
	#pragma omp single
	{
		printf("Level %d: number of threads in the team - %d\n", level, omp_get_num_threads());
	}
}